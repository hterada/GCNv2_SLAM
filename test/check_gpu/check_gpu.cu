#include <hip/hip_runtime.h>
#include <cstdio>

int main()
{
  int count = 0;
  auto result_count = hipGetDeviceCount(&count);
  if (hipSuccess != result_count) {
    std::printf("Error:result_count=%d\n", result_count );
    return -1;
  }
  if (count == 0) return -1;
  for (int device = 0; device < count; ++device)
  {
    hipDeviceProp_t prop;
    auto result = hipGetDeviceProperties(&prop, device);
    if (hipSuccess == result) {
      std::printf("%d.%d\n", prop.major, prop.minor);
    } else {
      std::printf("Error:result=%d\n", result);
    }
  }
  return 0;
}

